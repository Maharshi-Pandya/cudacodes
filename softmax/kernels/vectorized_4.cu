#include "hip/hip_runtime.h"
#include <assert.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <stdio.h>

#include "cuda_utils.cuh"

/*
This kernel implements an online softmax operation on a matrix of size (M, N).
The softmax operation is performed on the last dimension of the matrix.

How this works:
Instead of accessing shared memory and having sync barrier overhead, we will use warp-level primitives (then
block-level) for performing max and sum reductions. The benefit is: it is faster than shared
memory access and also does not need syncing since each warp (group of 32 threads) execute
an instuction parallely on GPU so no chance of race conditions.

We will also use vectorized loads and stores.
*/
__global__ void softmax_kernel_4(float* __restrict__ xd, float* __restrict__ resd, int M, int N) {
    // max and norm reduction will happen in shared memory (static)
    extern __shared__ float smem[];

    int row = blockIdx.x;
    int tid = threadIdx.x;
    if (row >= M) return;

    float* input_row = xd + row * N;
    float* output_row = resd + row * N;
    float local_max = -INFINITY;
    float local_norm = 0.0f;

    // cast as float4
    int n_float4s = N / 4;
    int tail = N % 4;
    float4* input_row_vec = reinterpret_cast<float4*>(input_row);
    float4* output_row_vec = reinterpret_cast<float4*>(output_row);
    float maxval = -INFINITY;

    #pragma unroll
    for (int i = tid; i < n_float4s; i += blockDim.x) {
        float4 elem = input_row_vec[i];

        maxval = fmaxf(maxval, elem.x);
        maxval = fmaxf(maxval, elem.y);
        maxval = fmaxf(maxval, elem.z);
        maxval = fmaxf(maxval, elem.w);
        if (maxval > local_max) {
            local_norm *= __expf(local_max - maxval);
            local_max = maxval;
        }
        local_norm += __expf(elem.x - maxval);
        local_norm += __expf(elem.y - maxval);
        local_norm += __expf(elem.z - maxval);
        local_norm += __expf(elem.w - maxval);
    }

    // handle extra row elements
    if (tail && tid < tail) {
        float val = input_row[n_float4s * 4 + tid];
        if (val > local_max) {
            local_norm *= __expf(local_max - val);
            local_max = val;
        }
        local_norm += __expf(val - local_max);
    }
    __syncthreads();

    // warp level reduction using XOR shuffle ('exchanges' the values in the threads)
    // note: if there are 256 threads in one block (8 warps of 32 threads each)
    // the following for loop reduces the value in all the 8 warps
    // the 8 warps contain the 8 maximum values of the 32 threads that reside in those warps
    // float val = smem[tid];
    blockReduceMax<float>(local_max, smem, -INFINITY);
    __syncthreads();

    // we got the global row max now
    float row_max = smem[0];
    __syncthreads();

    // each thread will have its own local_norm
    // we will store the corrected local_norm and reduce it
    // same reduction algorithm as above, but instead of max reduction
    // we do a sum reduction i.e. we accumulate the values
    float val = local_norm * expf(local_max - row_max);
    blockReduceSum<float>(val, smem, 0.0f);
    __syncthreads();

    float row_norm = smem[0];
    __syncthreads();

    // finally, compute softmax
    #pragma unroll
    for (int i = tid; i < n_float4s; i += blockDim.x) {
        float4 elem = input_row_vec[i];
        elem.x = __expf(elem.x - row_max) / row_norm;
        elem.y = __expf(elem.y - row_max) / row_norm;
        elem.z = __expf(elem.z - row_max) / row_norm;
        elem.w = __expf(elem.w - row_max) / row_norm;

        output_row_vec[i] = elem;
    }
    // write tail elements
    if (tail && tid < tail)
    {
        float val = input_row[n_float4s * 4 + tid];
        output_row[n_float4s * 4 + tid] = __expf(val - row_max) / row_norm;
    }
}

/*
Runs the online softmax kernel: `id = 4`
*/
float run_kernel_4(float* __restrict__ matd, float* __restrict__ resd, int M, int N) {
    // grid size and block size for this kernel
    // change as necessary
    dim3 block_size(1024);
    dim3 grid_size(M);

    int warp_size = 32;
    size_t smem_size = CEIL_DIV(block_size.x, warp_size) * sizeof(float);

    hipEvent_t start, stop;
    CUDA_CHECK(hipEventCreate(&start));
    CUDA_CHECK(hipEventCreate(&stop));
    float ms = 0.f;

    CUDA_CHECK(hipEventRecord(start));
    softmax_kernel_4<<<grid_size, block_size, smem_size>>>(matd, resd, M, N);
    CUDA_CHECK(hipEventRecord(stop));
    CUDA_CHECK(hipEventSynchronize(stop));
    CUDA_CHECK(hipEventElapsedTime(&ms, start, stop));
    printf(">> Kernel execution time: %f ms\n", ms);

    CUDA_CHECK(hipEventDestroy(start));
    CUDA_CHECK(hipEventDestroy(stop));

    return ms;
}