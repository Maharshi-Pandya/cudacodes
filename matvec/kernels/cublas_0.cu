#include "hip/hip_runtime.h"
#include <hipblas.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <stdio.h>

#include "utils.cuh"

/*
CuBLAS matrix vector multiplication for the baseline scores.
We simply run the Sgemv function that cuBLAS provides.
*/
void run_kernel_cublas_sgemv(float* __restrict__ matd, float* __restrict__ vecd, float* __restrict__ resd, int M, int N) {
    hipEvent_t start, stop;
    CUDA_CHECK(hipEventCreate(&start));
    CUDA_CHECK(hipEventCreate(&stop));
    float ms = 0.0f;

    // create cublas handle
    hipblasHandle_t handle;
    hipblasCreate(&handle);

    // Sgemv: y = (alpha * A * x) + (beta * y)
    float alpha = 1.0f, beta = 0.0f;
    hipEventRecord(start);
    hipblasSgemv(handle, HIPBLAS_OP_N, M, N, &alpha, matd, M, vecd, 1, &beta, resd, 1);
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&ms, start, stop);
    float gflops = compute_gflops(M, N, ms);
    printf(">> CuBLAS sgemv execution time: %f ms\n", ms);
    printf(">> CuBLAS sgemv (GFLOPS): %f\n", gflops);
    printf(">> Theoretical max (GFLOPS): %f\n", THEORETICAL_MAX_GFLOPS);
    printf(">> Maximum memory bandwidth: %f GB/s\n", THEORETICAL_MAX_MEMORY_BANDWIDTH);
    printf(">> CuBLAS sgemv achieves %f %% of peak GFLOPS\n", compute_peak_gflops(gflops));
    printf(">> CuBLAS sgemv achieves %f %% of peak Memory Bandwidth\n", compute_peak_memory_bandwidth(M, N, ms));

    hipblasDestroy(handle);
    CUDA_CHECK(hipEventDestroy(start));
    CUDA_CHECK(hipEventDestroy(stop));
}
