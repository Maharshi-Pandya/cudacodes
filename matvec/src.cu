#include <hipblas.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <stdio.h>

#include "coalesced_warp_2.cuh"
#include "coalesced_warpblock_3.cuh"
#include "cublas_0.cuh"
#include "naive_1.cuh"
#include "utils.cuh"

int main() {
    int M = 4096;
    int N = 4096;

    size_t matsize = M * N;  // (M, N)
    size_t vecsize = N;      // (N, 1)
    size_t mat_totalsize = matsize * sizeof(float);
    size_t vec_totalsize = vecsize * sizeof(float);

    // allocate host
    float *mat = (float *)malloc(mat_totalsize);
    float *vec = (float *)malloc(vec_totalsize);
    float *res = (float *)malloc(M * sizeof(float));

    for (size_t i = 0; i < matsize; i++) {
        mat[i] = random_normal_clamped(-10.f, 10.f);
        // hacky way to init the vector as well
        if (i < vecsize) {
            vec[i] = random_normal_clamped(-10.f, 10.f);
        }
    }

    hipEvent_t start, stop;
    CUDA_CHECK(hipEventCreate(&start));
    CUDA_CHECK(hipEventCreate(&stop));
    float ms = 0.0f;

    // allocate device
    float *matd, *vecd, *resd;
    hipEventRecord(start);
    CUDA_CHECK(hipMalloc((void **)&matd, mat_totalsize));
    CUDA_CHECK(hipMalloc((void **)&vecd, vec_totalsize));
    CUDA_CHECK(hipMalloc((void **)&resd, M * sizeof(float)));
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&ms, start, stop);
    printf(">> GPU allocation time: %f ms\n", ms);

    // copy host to device
    hipEventRecord(start);
    CUDA_CHECK(hipMemcpy(matd, mat, mat_totalsize, hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemcpy(vecd, vec, vec_totalsize, hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemcpy(resd, res, M * sizeof(float), hipMemcpyHostToDevice));
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&ms, start, stop);
    printf(">> Host to device transfer time: %f ms\n", ms);

    run_kernel_coalesced_warpblock_sgmev(matd, vecd, resd, M, N);

    // copy device to host
    hipEventRecord(start);
    CUDA_CHECK(hipMemcpy(res, resd, M * sizeof(float), hipMemcpyDeviceToHost));
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&ms, start, stop);
    printf(">> Device to host transfer time: %f ms\n", ms);

    // cleanup
    hipFree(matd);
    hipFree(vecd);
    hipFree(resd);
    free(mat);
    free(vec);
    free(res);
}